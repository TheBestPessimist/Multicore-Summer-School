#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <Windows.h>
			/*for (int p=0; p<poza1->Width; p+=1)
				{
					for (int pp=0; pp<poza1->Height; pp+=1)
					{
						if (p==pp)
							puncteStarDP.Add(Point(
							p,
							pp								
							));
					}
				}//diagonala principala

			for (int s=0; s<poza1->Width; s+=1)
				{
					for (int ss=0; ss<poza1->Height; ss+=1)
					{
						if (ss==poza1->Height+1-s)
							puncteStarDS.Add(Point(
							s,
							ss								
							));
					}
				}//diagonala secundara

			for (int v=0; v<poza1->Width; v+=1)
				{
					for (int vv=0; vv<poza1->Height/2; vv+=1)
					{
						puncteStarDV.Add(Point(
							v,
							vv=poza1->Width/2								
							));
					}
				}//verticala mijloc

			for (int o=0; o<poza1->Width/2; o+=1)
				{
					for (int oo=0; oo<poza1->Height; oo+=1)
					{
						
							puncteStarDO.Add(Point(
							o=poza1->Height/2,
							oo								
							));
					}
				}//orizontala mijloc*/


//------------------------------------DLLEXPORTS----------------------------------------------	
extern "C"
{
	__declspec(dllexport) void pixelStarCallF(int *,int *, int *, int *, int *,int *, int*, int*, int , int*, int*, int*, int*);
	__declspec(dllexport) void randCallF(int, int , int *, int *);
}
//------------------------------------ENDDLLEXPORTS-------------------------------------------				
//------------------------------------KERNELS-------------------------------------------------
__device__ int my_rand()
{
  int r[100000];
  int i;
  int seed = 1354681;
  r[0] = seed;
  for (i=1; i<31; i++) {
	 r[i] = (16807LL * r[i-1]) % 2147483647;
	 if (r[i] < 0) {
		r[i] += 2147483647;
	 }
  }
  for (i=31; i<34; i++) {
	 r[i] = r[i-31];
  }
  for (i=34; i<344; i++) {
	 r[i] = r[i-31] + r[i-3];
  }
	i=334;
	r[i] = r[i-31] + r[i-3];
  i=((unsigned int)r[i])>>1;
  return i;
}
__global__ void pixelStar_kernel(int *DPx, int *DPy, int *DSx, int *DSy, int *DVx,int *DVy, int *DOx,int *DOy, int W, int *k1, int *k2, int *k3, int *k4)
{
	int 	i=blockIdx.x * blockDim.x + threadIdx.x;
	int	j=blockIdx.y * blockDim.y + threadIdx.y;
	if(i==0&&j==0){
		*k1=0, *k2=0, *k3=0, *k4=0;
	}
	 __syncthreads();
	if(i<(W-1) && j<(W-1))
	{
		if(i==j)
		{
			DPx[i]=i;
			DPy[j]=j;
		}
		//__syncthreads();
	   if(j==(W-1-i))
	   {
	   	DSx[i]=W+1-i;
		   DSy[i]=j;
   	}
		//__syncthreads();
		if(j==W/2)
		{
			DVx[i]=i;
			DVy[i]=j;
		}
		//__syncthreads();
		if(i==W/2)
		{
			DOx[j]=i;
			DOy[j]=j;
		}
	}
}
__global__ void pixalRandom_kernel(int W, int *PRx, int *PRy, int tbv)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if(i<tbv)
	{
		PRx[i] = my_rand() % (W-1) +1 ;
		PRy[i] = my_rand() % (W-1) +1 ;
	}
}
//------------------------------------ENDKERNELS----------------------------------------------

//------------------------------------KERNELCALLS---------------------------------------------
void pixelStarCallF(int *DPx, int *DPy, int *DSx, int *DSy, int *DVx,int *DVy, int*DOx,int*DOy, int W, int*k1, int*k2, int*k3, int*k4)
{
	int *dk1, *dk2, *dk3, *dk4;
	int *dDPx, *dDSx, *dDVx, *dDOx;
	int *dDPy, *dDSy, *dDVy, *dDOy;
	hipMalloc((void**)&dk1, sizeof(int));
	hipMalloc((void**)&dk2, sizeof(int));
	hipMalloc((void**)&dk3, sizeof(int));
	hipMalloc((void**)&dk4, sizeof(int));
	hipMalloc((void**)&dDPx, W*sizeof(int));
	hipMalloc((void**)&dDPy, W*sizeof(int));
	hipMalloc((void**)&dDSx, W*sizeof(int));
	hipMalloc((void**)&dDSy, W*sizeof(int));
	hipMalloc((void**)&dDVx, W*sizeof(int));
	hipMalloc((void**)&dDVy, W*sizeof(int));
	hipMalloc((void**)&dDOx, W*sizeof(int));
	hipMalloc((void**)&dDOy, W*sizeof(int));
	
   int bdim=W/32+1;
	dim3 blocks(bdim,bdim,1);
	dim3 threads(32,32,1);
	pixelStar_kernel<<<blocks, threads>>>(dDPx, dDPy, dDSx,dDSy, dDVx,dDVy, dDOx,dDOy, W, dk1, dk2, dk3, dk4);
	hipDeviceSynchronize();
	
	hipMemcpy(DPx, dDPx, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DPy, dDPy, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DSx, dDSx, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DSy, dDSy, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DVx, dDVx, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DVy, dDVy, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DOx, dDOx, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(DOy, dDOy, W*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(k1, dk1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(k2, dk2, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(k3, dk3, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(k4, dk4, sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dk1);
	hipFree(dk2);
	hipFree(dk3);
	hipFree(dk4);
	hipFree(dDPx);
	hipFree(dDPy);
	hipFree(dDSx);
	hipFree(dDSy);
	hipFree(dDVx);
	hipFree(dDVy);
	hipFree(dDOx);
	hipFree(dDOy);
}
void randCallF(int W,int tbv, int *PRx, int *PRy)
{
	int *dPRx, *dPRy;
	hipMalloc((void**)&dPRx, tbv*sizeof(int));
	hipMalloc((void**)&dPRy, tbv*sizeof(int));
	
	pixalRandom_kernel<<<(tbv/1024)+1,1024>>>( W, dPRx, dPRy, tbv); 
	
	hipMemcpy(PRx, dPRx, tbv*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(PRy, dPRy, tbv*sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dPRx);
	hipFree(dPRy);
}
//------------------------------------ENDKERNELCALLS------------------------------------------